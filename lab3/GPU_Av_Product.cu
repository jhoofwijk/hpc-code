#include "hip/hip_runtime.h"
/*****************************************************************************
This function finds the product of Matrix A and vector V
*****************************************************************************/

// ****************************************************************************************************************************************************/
// parallelization method for the Matrix-vector multiplication as follows: 

// each thread handle a multiplication of each row of Matrix A and vector V;

// The share memory is limited for a block, instead of reading an entire row of matrix A or vector V from global memory to share memory, 
// a square submatrix of A is shared by a block, the size of square submatrix is BLOCK_SIZE*BLOCK_SIZE; Thus, a for-loop is used to
// handle a multiplication of each row of Matrix A and vector V step by step. In eacg step, two subvectors with size BLOCK_SIZE is multiplied.
//*****************************************************************************************************************************************************/


__global__ void Av_Product(float* g_MatA, float* g_VecV, float* g_VecW, int N)
{
    // Block index
    int bx = blockIdx.x;

    // Thread index
    int tx = threadIdx.x;

    int aBegin = N * BLOCK_SIZE * bx;

    int aEnd   = aBegin + N - 1;
    int step  = BLOCK_SIZE;

    int bBegin = 0;//BLOCK_SIZE * bx;
    int bIndex=0;
    int aIndex =0;
    float Csub = 0;

    for (int a = aBegin, b = bBegin;
         a <= aEnd;
         a += step, b += step)
    {

        __shared__ float As[BLOCK_SIZE*BLOCK_SIZE];

        __shared__ float bs[BLOCK_SIZE];
        

        for (int aa = 0; aa < BLOCK_SIZE;aa+= 1)
        {
            aIndex = a+tx+aa*N;
            if( aIndex < N*N)
        	    As[tx+aa*BLOCK_SIZE] = g_MatA[aIndex];
		        else
        	    As[tx+aa*BLOCK_SIZE] = 0;
        }

        bIndex = b+tx;
   	    if(bIndex<N)   
		      bs[tx] = g_VecV[bIndex];
	      else
		      bs[tx] = 0;

        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            Csub += As[k+tx*BLOCK_SIZE] * bs[k];
        }//}
        __syncthreads();
    }

    g_VecW[ BLOCK_SIZE * bx + tx] = Csub;
}

